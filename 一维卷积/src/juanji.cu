#include <iostream>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <time.h>
__global__ void neiji(int *d_a,int *d_b,int *d_c,int *d_data,int n, int m,int IJdx)
{
	int temp = 0;
	int INdx = threadIdx.x;
	d_data[IJdx * m + INdx] = d_a[IJdx + INdx] * d_b[INdx];
	__syncthreads();
	for(int i = 0; i < m; ++i)
	{
		temp += d_data[IJdx * m + i];
	}
	d_c[IJdx] = temp;
}
__global__ void juanji(int *d_a,int *d_b,int *d_c,int *d_data,int n, int m)
{
	int IJdx = threadIdx.x;
	neiji<<<1,m>>>(d_a,d_b,d_c,d_data,n,m,IJdx);
	__syncthreads();
}
int main()
{
	int blag = 1;
	int m = 0, n = 0;
	do{
		std::cout << "请输入一位向量的长度:"<< std::endl;
		std::cin >> n;
		std::cout << "请输入卷积因子长度:" <<std::endl;
		std::cin >> m;
		if(m > n)
		{
			std::cout << "你输入的数据不合法，请重新输入！"<< std::endl;
		}else{
			blag = 0;
		}
	}while(blag);
	/*****申请主机内存*******/
	int *h_a,*h_b,*h_c;
	h_a = (int*)malloc(sizeof(int) * n);
	h_b = (int*)malloc(sizeof(int) * m);
	h_c = (int*)malloc(sizeof(int) * (n - m + 1));
	/******生成随机数据*******/
	srand(time(NULL));
	for(int i = 0; i < n; ++i)
	{
		h_a[i] = rand() % 11;
		printf("h_a[%d] = %d\t",i,h_a[i]);
	}
	printf("\n");
	for(int i = 0; i < m; ++i)
	{
		h_b[i] = rand() % 11;
		printf("h_b[%d] = %d\t",i,h_b[i]);
	}
	printf("\n");

	/******申请设备内存*************/
	int *d_a,*d_b,*d_c,*d_data;
	hipMalloc((void**)&d_a,sizeof(int) * n);
	hipMalloc((void**)&d_b,sizeof(int) * m);
	hipMalloc((void**)&d_c,sizeof(int) * (n - m + 1));
	hipMalloc((void**)&d_data,sizeof(int) * m * (n - m + 1));
	/******主机内存数据复制到设备内存中********/
	hipMemcpy(d_a,h_a,sizeof(int) * n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(int) * m,hipMemcpyHostToDevice);

	/*******启动核函数*******/
	juanji<<<1,n - m + 1>>>(d_a,d_b,d_c,d_data,n,m);

	/*******设备内存数据复制到主机内存中********/
	hipMemcpy(h_c,d_c,sizeof(int) * (n - m + 1),hipMemcpyDeviceToHost);

	for(int i = 0; i < n - m + 1; ++i)
	{
		printf("h_c[%d] = %d\t",i,h_c[i]);
	}
	printf("\n");

	/********释放内存********/
	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	std::cout << "运行结束" << std::endl;
	return 0;
}
