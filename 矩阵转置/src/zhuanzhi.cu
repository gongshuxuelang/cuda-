#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>

__global__ void zhuanzhi(int *d_A,int *d_B,int n)
{
	int ITdx = threadIdx.x;
	int IBdx = blockIdx.x;
	d_B[ITdx * n + IBdx] = d_A[IBdx * n + ITdx];
}
int main()
{
	int blag = 1;//设置标识位
	int n = 0;
	srand(time(NULL));
	/*******判断输入数据是否合法*****/
	do{
		std::cout << "请输入矩阵维度:" << std::endl;
		std::cin >> n;
		if(n <= 0)
		{
			std::cout << "你输入的矩阵维度有误,请重新输入!" << std::endl;
		}else{
			blag = 0;
		}
	}while(blag);

	/******申请主机内存*****/
	int *H_A,*H_B;
	H_A = (int*)malloc(sizeof(int) * n * n);
	H_B = (int*)malloc(sizeof(int) * n * n);

	/*******申请设备内存**************/
	int *d_A,*d_B;

	hipMalloc((void**)&d_A,sizeof(int) * n * n);
	hipMalloc((void**)&d_B,sizeof(int) * n * n);

	/****初始化二维数组的数值*******/
	for(int i = 0; i < n * n; ++i)
	{
		H_A[i] = rand() % 11;
		H_B[i] = 0;
	}


	/********主机内存数据复制到设备内存中***********/
	hipMemcpy(d_A,H_A,sizeof(int) * n * n,hipMemcpyHostToDevice);

	/*******启动核函数********/
	zhuanzhi<<<n,n>>>(d_A,d_B,n);
	hipDeviceSynchronize();

	/*******设备内存数据复制到主机内存中*********/
	hipMemcpy(H_B,d_B,sizeof(int) * n *n,hipMemcpyDeviceToHost);

	/****打印矩阵和转置矩阵*******/
	std::cout << "CPU 输出" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			std::cout << "H_A[" << i << "][" << j << "] =" << H_A[n * i + j] <<"  ";
		}
		std::cout << std::endl;
	}

	std::cout << "转置结果:" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			std::cout << "H_B[" << i << "][" << j << "] =" << H_B[n * i + j] <<"  ";
		}
		std::cout << std::endl;
	}

	/********释放内存*********/
	free(H_A);
	free(H_B);
	hipFree(d_A);
	hipFree(d_B);
	std::cout << "运行结束" << std::endl;
	return 0;
}
