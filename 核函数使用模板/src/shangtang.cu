#include <iostream>
#include <hip/hip_runtime.h>
#include <ctime>
#include <cstdlib>

template<typename T>
__global__ void add(T *d_a,T *d_b,T *d_c,int n)
{
	int idx = threadIdx.x;
	d_c[idx] = d_a[idx] + d_b[idx];
}

int main()
{
	int n = 0;
	int blag = 1;//标志位
	do{
		std::cout << "请输入数组的长度:" << std::endl;
		std::cin >> n;
		if(n <= 0)
		{
			std::cout << "你输入的数组长度为为正数,请重新输入:" << std::endl;
		}else
		{
			blag = 0;
		}
	}while(blag);

	/******申请主机内存******/
	double * h_a = (double*)malloc(sizeof(double) * n);
	double * h_b = (double*)malloc(sizeof(double) * n);
	double * h_c = (double*)malloc(sizeof(double) * n);
	/******主机内存赋值********/
	srand(time(NULL));
	for(int i = 0; i < n; ++i)
	{
		h_a[i] = rand() % 101 / 10.0;
		h_b[i] = rand() % 101 / 10.0;
	}
	/******申请设备内存**********/
	double *d_a,*d_b,*d_c;
	hipMalloc((void**)&d_a,sizeof(double) * n);
	hipMalloc((void**)&d_b,sizeof(double) * n);
	hipMalloc((void**)&d_c,sizeof(double) * n);

	/******主机内存数据复制到设备内存********/
	hipMemcpy(d_a,h_a,sizeof(double) * n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(double) * n,hipMemcpyHostToDevice);

	/*****启动核函数********/
	add<double><<<1,n>>>(d_a,d_b,d_c,n);

	/*****设备内存数据复制到主机内存*********/
	hipMemcpy(h_c,d_c,sizeof(double) * n,hipMemcpyDeviceToHost);
	for(int i = 0; i < n; ++i)
	{
		std::cout << "h_a[" << i << "] = " << h_a[i] << "  ";
	}
	std::cout << std::endl;	for(int i = 0; i < n; ++i)
	{
		std::cout << "h_b[" << i << "] = " << h_b[i] << "  ";
	}
	std::cout << std::endl;
	for(int i = 0; i < n; ++i)
	{
		std::cout << "h_c[" << i << "] = " << h_c[i] << "  ";
	}
	std::cout << std::endl;
	/*******释放设备内存*****/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*****释放主机内存*****/
	free(h_a);
	free(h_b);
	free(h_c);

	std::cout << "运行结束!" << std::endl;

	return 0;
}
