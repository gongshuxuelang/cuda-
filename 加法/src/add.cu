#include <hip/hip_runtime.h>
#include <iostream>

__global__ void add(int *d_a,int *d_b,int *d_c,int n)
{
	int idx = threadIdx.x;
	d_c[idx] = d_a[idx] + d_b[idx];
}
int main()
{
	int n = 0;
	int blag = 1;//标志位
	do{
		std::cout << "请输入数组的长度:" << std::endl;
		std::cin >> n;
		if(n <= 0)
		{
			std::cout << "你输入的数组长度为为正数,请重新输入:" << std::endl;
		}else
		{
			blag = 0;
		}
	}while(blag);

	/******申请主机内存******/
	int * h_a = (int*)malloc(sizeof(int) * n);
	int * h_b = (int*)malloc(sizeof(int) * n);
	int * h_c = (int*)malloc(sizeof(int) * n);
	/******主机内存赋值********/
	for(int i = 0; i < n; ++i)
	{
		h_a[i] = i + 1;
		h_b[i] = i + 3;
	}
	/******申请设备内存**********/
	int *d_a,*d_b,*d_c;
	hipMalloc((void**)&d_a,sizeof(int) * n);
	hipMalloc((void**)&d_b,sizeof(int) * n);
	hipMalloc((void**)&d_c,sizeof(int) * n);

	/******主机内存数据复制到设备内存********/
	hipMemcpy(d_a,h_a,sizeof(int) * n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(int) * n,hipMemcpyHostToDevice);

	/*****启动核函数********/
	add<<<1,n>>>(d_a,d_b,d_c,n);

	/*****设备内存数据复制到主机内存*********/
	hipMemcpy(h_c,d_c,sizeof(int) * n,hipMemcpyDeviceToHost);

	for(int i = 0; i < n; ++i)
	{
		std::cout << "h_c[" << i << "] = " << h_c[i] << "  ";
	}
	std::cout << std::endl;
	/*******释放设备内存*****/
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	/*****释放主机内存*****/
	free(h_a);
	free(h_b);
	free(h_c);

	std::cout << "运行结束!" << std::endl;
	return 0;
}
