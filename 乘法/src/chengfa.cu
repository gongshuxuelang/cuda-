#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>

__global__ void zhuanshi(int *d_b,int *d_bt,int n)
{
	int ITdx = threadIdx.x;
	int IBdx = blockIdx.x;

	d_bt[ITdx * n + IBdx] = d_b[IBdx * n + ITdx];
}
__global__ void neiji(int *d_a,int *d_bt,int *d_c,int *d_data,int ICTdx,int ICBdx,int n)
{
	/*
	int INTdx = threadIdx.x;
	int i = 2, j = 1;

	d_data[INTdx] = d_a[ICTdx * n + INTdx] * d_bt[ICBdx * n + INTdx];

	__syncthreads();
	while(i <= n)
	{
		if(INTdx % 2 == 0)
		{
			d_data[INTdx] += d_data[INTdx + j];
		}
		i *= 2;
		j *= 2;
	}
	d_c[ICTdx * n + ICBdx] = d_data[0];
*/
}
__global__ void chengfa(int *d_a,int *d_bt,int *d_c,int *d_data,int n)
{
/*
	int ICTdx = threadIdx.x;
	int ICBdx = blockIdx.x;
	neiji<<<1,n>>>(d_a,d_bt,d_c,d_data,ICTdx,ICBdx,n);
	__syncthreads();
*/
}

int main()
{
	int blag = 1;//标志位
	int n = 0;
	/******判断输入数据是否合法************/
	do{
		std::cout << "请输入矩阵的维度:" << std::endl;
		std::cin >> n;
		if(n <= 0)
		{
			std::cout << "你输入的矩阵维度有误,请重新输入!" << std::endl;
		}else{
			blag = 0;
		}
	}while(blag);

	/*******申请主机内存*********/
	int *h_a = (int*)malloc(sizeof(int) * n * n);
	int *h_b = (int*)malloc(sizeof(int) * n * n);
	int *h_c = (int*)malloc(sizeof(int) * n * n);
	int *h_bt = (int*)malloc(sizeof(int) * n * n);
	/*******初始化主机内存数据********/
	srand(time(NULL));//设置随机数值
	for(int i = 0; i < n * n; ++i)
	{
		h_a[i] = rand() % 11;
		h_b[i] = rand() % 11;
		h_c[i] = 0;
		h_bt[i] = 0;
	}

	/*******申请设备内存*******/
	int *d_a,*d_b,*d_c,*d_bt,*d_data;
	hipMalloc((void**)&d_a,sizeof(int) * n * n);
	hipMalloc((void**)&d_b,sizeof(int) * n * n);
	hipMalloc((void**)&d_c,sizeof(int) * n * n);
	hipMalloc((void**)&d_bt,sizeof(int) * n * n);
	hipMalloc((void**)&d_data,sizeof(int)*n);

	/******主机内存数据复制到设备内存中************/
	hipMemcpy(d_a,h_a,sizeof(int) * n * n,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,sizeof(int) * n * n,hipMemcpyHostToDevice);
	std::cout << "测试点" << std::endl;
	/*******执行核函数******/
	zhuanshi<<<n,n>>>(d_b,d_bt,n);
	chengfa<<<n,n>>>(d_a,d_bt,d_c,d_data,n);

	/*****设备内存数据复制到主机内存中*****/
	hipMemcpy(h_bt,d_bt,sizeof(int) * n * n,hipMemcpyDeviceToHost);
	hipMemcpy(h_c,d_c,sizeof(int) * n * n,hipMemcpyDeviceToHost);

	std::cout << "CPU内存数据h_a:" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			printf("h_a[%d][%d] = %d\t",i,j,h_a[n * i + j]);
		}
		printf("\n");
	}
	std::cout << "CPU内存数据h_b:" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			printf("h_b[%d][%d] = %d\t",i,j,h_b[n * i + j]);
		}
		printf("\n");
	}
	std::cout << "CPU内存数据h_bt:" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			printf("h_bt[%d][%d] = %d\t",i,j,h_bt[n * i + j]);
		}
		printf("\n");
	}
	std::cout << "GPU内存数据:" << std::endl;
	for(int i = 0; i < n; ++i)
	{
		for(int j = 0; j < n; ++j)
		{
			printf("h_c[%d][%d] = %d\t",i,j,h_c[n * i + j]);
		}
		printf("\n");
	}

	/*******释放内存*********/
	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);

	std::cout << "运行结束" << std::endl;
	return 0;
}
