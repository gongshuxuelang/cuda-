#include <hip/hip_runtime.h>
#include <iostream>

class CUDAdem
{
public:
	__device__ void add(int igdx)
	{
		printf("hello GPU = %d\n",igdx);
	}
};
__global__ void add()
{
	int igdx = threadIdx.x;
	CUDAdem cdmo;
	cdmo.add(igdx);
}
int main()
{

	add<<<1,4>>>();
	hipDeviceReset();
	printf("hello world!\n");
	return 0;
}
