#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <time.h>

__global__ void add(int * d_a,int *d_b,int n)
{
	int idx = threadIdx.x;
	int i = 2,j = 1;

	do{
		if(idx % i == 0)
		d_a[idx] += d_a[idx + j];
		i *= 2;
		j *= 2;
	}while(n/=2);
	d_b[0] = d_a[0];

}
int main()
{
	int blag = 1;
	int n = 0;
	do{
		std::cout << "请输入数据长度:" << std::endl;
		std::cin >> n;
		if(n <= 0)
		{
			std::cout << "你输入的数据长度不合法,请重新输入!" << std::endl;
		}else{
			blag = 0;
		}
	}while(blag);

	srand(time(NULL));

	int *h_a = (int*)malloc(sizeof(int) * n);
	int *h_b = (int*)malloc(sizeof(int));
	for(int i = 0; i < n; ++i)
	{
		h_a[i] = rand() % 11;
		printf("h_a[%d] = %d\t",i,h_a[i]);
	}
	printf("\n");

	int *d_a = NULL;
	int *d_b = NULL;
	hipMalloc((void**)&d_a,sizeof(int) * n);
	hipMalloc((void**)&d_b,sizeof(int));

	hipMemcpy(d_a,h_a,sizeof(int) * n,hipMemcpyHostToDevice);

	add<<<1,n>>>(d_a,d_b,n);
	hipMemcpy(h_b,d_b,sizeof(int),hipMemcpyDeviceToHost);

	printf("h_b = %d\n",*h_b);

	free(h_a);
	free(h_b);
	hipFree(d_a);
	hipFree(d_b);

	printf("运行结束!\n");
	return 0;
}
