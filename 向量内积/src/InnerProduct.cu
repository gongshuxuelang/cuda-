#include <hip/hip_runtime.h>
#include <iostream>

__global__ void InnerProduct(int *d_a,int *d_b,int *d_c,int n)
{
	int Idx = threadIdx.x;
	int IdxMax = blockDim.x;
	do{
		d_c[Idx] = d_a[Idx] * d_b[Idx];
		IdxMax += n;
	}while(IdxMax < n);
	d_c[Idx] = d_a[Idx] * d_b[Idx];
	__syncthreads();

	int i = 2, j = 1;
	while(i <= n)
	{
		if(Idx % 2 == 0)
		{
			d_c[Idx] += d_c[Idx + j];
		}
		i *= 2;
		j *= 2;
	}
}
int main()
{
	int blag = 1; //标志位
	int n = 0; //数据大小

	do{
		std::cout << "请输入数据大小:" << std::endl;
		std::cin >> n;
		if(n < 0)
		{
			std::cout << "你输入的数据是错误的,请重新输入!" << std::endl;
		}else
		{
			blag = 0;
		}
	}while(blag);

	/*********申请主机内存*************/
	int *h_a,*h_b,*h_c;
	int nByte = sizeof(int) * n;
	h_a = (int*)malloc(nByte);
	h_b = (int*)malloc(nByte);
	h_c = (int*)malloc(nByte);

	/********申请设备内存 *************/
	int *d_a,*d_b,*d_c;
	hipMalloc((void**)&d_a,nByte);
	hipMalloc((void**)&d_b,nByte);
	hipMalloc((void**)&d_c,nByte);

	/******给主机内存赋值**************/
	for(int i = 0; i < n; ++i)
	{
		h_a[i] = i + 1;
		h_b[i] = i + 3;
	}

	/******主机内存数据复制到设备内存中***/
	hipMemcpy(d_a,h_a,nByte,hipMemcpyHostToDevice);
	hipMemcpy(d_b,h_b,nByte,hipMemcpyHostToDevice);

	/*****执行核函数*****/
	InnerProduct<<<1,n>>>(d_a,d_b,d_c,n);

	/*******设备数据复制到主机内存**************/
	hipMemcpy(h_c,d_c,nByte,hipMemcpyDeviceToHost);

	/**********输出结果***********/

	std::cout << "h_c = " << h_c[0] << std::endl;

	/******释放内存*****/
	free(h_a);
	free(h_b);
	free(h_c);
	hipFree(d_a);
	hipFree(d_b);
	hipFree(d_c);
	std::cout << "运行完毕!" << std::endl;
	return 0;
}
